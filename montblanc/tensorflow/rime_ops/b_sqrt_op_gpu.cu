
#include <hip/hip_runtime.h>
#if GOOGLE_CUDA

#include "b_sqrt_op_gpu.cuh"

namespace montblanc {
namespace bsqrt {

REGISTER_KERNEL_BUILDER(
    Name("RimeBSqrt")
    .Device(tensorflow::DEVICE_GPU)
    .HostMemory("ref_freq")
    .TypeConstraint<float>("FT")
    .TypeConstraint<tensorflow::complex64>("CT"),
    RimeBSqrt<GPUDevice, float, tensorflow::complex64>);

REGISTER_KERNEL_BUILDER(
    Name("RimeBSqrt")
    .Device(tensorflow::DEVICE_GPU)
    .HostMemory("ref_freq")
    .TypeConstraint<double>("FT")
    .TypeConstraint<tensorflow::complex128>("CT"),
    RimeBSqrt<GPUDevice, double, tensorflow::complex128>);

} // namespace bsqrt {
} // namespace montblanc {

#endif