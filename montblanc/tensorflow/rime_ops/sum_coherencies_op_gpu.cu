
#include <hip/hip_runtime.h>
#if GOOGLE_CUDA

#include "sum_coherencies_op_gpu.cuh"

namespace montblanc {
namespace sumcoherencies {

REGISTER_KERNEL_BUILDER(
    Name("RimeSumCoherencies")
    .Device(tensorflow::DEVICE_GPU)
    .HostMemory("src_lower")
    .HostMemory("src_upper")
    .TypeConstraint<float>("FT")
    .TypeConstraint<tensorflow::complex64>("CT"),
    RimeSumCoherencies<GPUDevice, float, tensorflow::complex64>);

REGISTER_KERNEL_BUILDER(
    Name("RimeSumCoherencies")
    .Device(tensorflow::DEVICE_GPU)
    .HostMemory("src_lower")
    .HostMemory("src_upper")
    .TypeConstraint<double>("FT")
    .TypeConstraint<tensorflow::complex128>("CT"),
    RimeSumCoherencies<GPUDevice, double, tensorflow::complex128>);

} // namespace sumcoherencies {
} // namespace montblanc {

#endif